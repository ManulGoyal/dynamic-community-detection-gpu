#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <vector>
#include <iostream>
#include <thrust/partition.h>
#include <fstream>
#include <getopt.h>
#include <sstream>
// #include <string.h>
#include "../louvain.h"

host_structures readInputData(char *fileName) {
	std::fstream file;
	file.open(fileName);
    int V, E;
	std::string s;
	do {
		std::getline(file, s);
	} while (s[0] == '%');
	std::istringstream stream(s);
    stream >> V >> V >> E;
	printf("Vertices: %d, Edges: %d\n", V, E);
    int v1, v2;
    float w;
    host_structures hostStructures;
	hostStructures.originalV = V;
	hostStructures.V = V;
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.vertexCommunity, V * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.communityWeight, V * sizeof(float), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edgesIndex, (V + 1) * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.originalToCommunity, V * sizeof(int), hipHostMallocDefault));

    std::vector<std::vector<std::pair<int, float>>> neighbours(V);
    // TODO: here is assumption that graph is undirected
    int aux = E;
    for (int i = 0; i < aux; i++) {
        file >> v1 >> v2 >> w;
        // v1--;
        // v2--;
		hostStructures.communityWeight[v1] += w;
        neighbours[v1].emplace_back(v2, w);
        if (v1 != v2) {
            E++;
			hostStructures.communityWeight[v2] += w;
            neighbours[v2].emplace_back(v1, w);
			hostStructures.M += w;
        }
		hostStructures.M += w;
    }
    hostStructures.M /= 2;
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edges, E * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.weights, E * sizeof(float), hipHostMallocDefault));
	hostStructures.E = E;
    int index = 0;
    for (int v = 0; v < V; v++) {
		hostStructures.edgesIndex[v] = index;
        for (auto & it : neighbours[v]) {
			hostStructures.edges[index] = it.first;
			hostStructures.weights[index] = it.second;
            index++;
        }
    }
	hostStructures.edgesIndex[V] = E;
    file.close();
	printf("Graph reading done\n");
    return hostStructures;
}

host_structures convertToHostStructures(Graph& gr) {
	// std::fstream file;
	// file.open(fileName);
    int V, E;
	// std::string s;
	// do {
	// 	std::getline(file, s);
	// } while (s[0] == '%');
	// std::istringstream stream(s);
    // stream >> V >> V >> E;
	V = gr.nb_nodes;
	E = gr.nb_links;
    int v1, v2;
    float w;
    host_structures hostStructures;
	hostStructures.originalV = V;
	hostStructures.V = V;
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.vertexCommunity, V * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.communityWeight, V * sizeof(float), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edgesIndex, (V + 1) * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.originalToCommunity, V * sizeof(int), hipHostMallocDefault));

	cout << "Point 1G" << endl;

    // std::vector<std::vector<std::pair<int, float>>> neighbours(V);

	// for(int i = 0; i < V; i++) {
	// 	pair<vector<int>::iterator, vector<long double>::iterator> p = gr.neighbors(i);//
	// 	int deg = gr.nb_neighbors(i);
	// 	for (int i=0 ; i<deg ; i++) {
	// 		int neigh  = *(p.first+i);//ith nighbor of node

	// 		// Manul: weight of edge from node to its i-th neighbor
	// 		long double neigh_w = (gr.weights.size()==0)?1.0L:*(p.second+i);//ith neighbor comm weight

	// 		hostStructures.communityWeight[i] += neigh_w;
    //     	neighbours[i].emplace_back(neigh, neigh_w);	

	// 		hostStructures.M += neigh_w;	
	// 	}
	// }

	for(int i = 0; i < V; i++) {
		hostStructures.communityWeight[i] = gr.weighted_degree(i);
	}

    hostStructures.M = (gr.total_weight) / 2;
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edges, E * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.weights, E * sizeof(float), hipHostMallocDefault));
	hostStructures.E = E;

	cout << "Point 2G" << endl;


	assert(E == gr.links.size());
	copy(gr.links.begin(), gr.links.end(), hostStructures.edges);
	if(gr.weights.empty()) {
		fill(hostStructures.weights, hostStructures.weights + hostStructures.E, 1.0f);
	} else {
		copy(gr.weights.begin(), gr.weights.end(), hostStructures.weights);
	}
	*(hostStructures.edgesIndex) = 0;
	copy(gr.degrees.begin(), gr.degrees.end(), hostStructures.edgesIndex + 1);

	cout << "Point 3G" << endl;

    // int index = 0;
    // for (int v = 0; v < V; v++) {
	// 	hostStructures.edgesIndex[v] = index;
    //     for (auto & it : neighbours[v]) {
	// 		hostStructures.edges[index] = it.first;
	// 		hostStructures.weights[index] = it.second;
    //         index++;
    //     }
    // }
	// hostStructures.edgesIndex[V] = E;
    // file.close();
    return hostStructures;
}

void init_partition(host_structures& hostStructures, int* comm_size, char *filename) {
	std::fstream file;
	file.open(filename);

	float* vertexEdgeWeights = (float*) malloc(hostStructures.V*sizeof(float));
	for(int j = 0; j < hostStructures.V; j++) {
		vertexEdgeWeights[j] = hostStructures.communityWeight[j];
		hostStructures.communityWeight[j] = 0;
	}

	cout << "Point 4G" << endl;


	for (int i = 0; i < hostStructures.V; i++)
	{
		int node, comm; file >> node >> comm;
		
		// copy(hostStructures.communityWeight, hostStructures.communityWeight + hostStructures.V, vertexEdgeWeights);
		// fill(hostStructures.communityWeight, hostStructures.communityWeight + hostStructures.V, 0);
		hostStructures.vertexCommunity[node] = comm;
		hostStructures.communityWeight[comm] += vertexEdgeWeights[node];
		hostStructures.originalToCommunity[node] = comm;
		comm_size[comm] += 1;
	}
	free(vertexEdgeWeights);
	printf("Init partition done\n");
}

void init_partition(host_structures& hostStructures, int* comm_size, std::vector<int>& n2c) {
	cout << "INIT PART starts" << endl;
	
	assert(n2c.size() == hostStructures.V);

	float* vertexEdgeWeights = (float*) malloc(hostStructures.V*sizeof(float));

	cout << "After malloc" << endl;
	
	for(int j = 0; j < hostStructures.V; j++) {
		vertexEdgeWeights[j] = hostStructures.communityWeight[j];
		hostStructures.communityWeight[j] = 0;
	}

	cout << "Point 1INIT" << endl;

	
	for (int i = 0; i < hostStructures.V; i++)
	{
		int node = i, comm = n2c[i];
		
		// copy(hostStructures.communityWeight, hostStructures.communityWeight + hostStructures.V, vertexEdgeWeights);
		// fill(hostStructures.communityWeight, hostStructures.communityWeight + hostStructures.V, 0);
		hostStructures.vertexCommunity[node] = comm;
		hostStructures.communityWeight[comm] += vertexEdgeWeights[node];
		// hostStructures.originalToCommunity[node] = comm;		// Manul: TODO: Not needed
		comm_size[comm] += 1;
	}

	free(vertexEdgeWeights);
	printf("Init partition done\n");
}

void copyStructures(host_structures& hostStructures, device_structures& deviceStructures,
					aggregation_phase_structures& aggregationPhaseStructures) {
	// copying from deviceStructures to hostStructures
	int V = hostStructures.V, E = hostStructures.E;
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communityWeight, V * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.weights, E * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edgesIndex, (V + 1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalToCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexEdgesSum, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.newVertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.V, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.E, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalV, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communitySize, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.partition, V * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&deviceStructures.toOwnCommunity, V * sizeof(int)));


	thrust::fill(thrust::device, deviceStructures.communitySize, deviceStructures.communitySize + V, 1);
	thrust::sequence(thrust::device, deviceStructures.vertexCommunity, deviceStructures.vertexCommunity + V, 0);
	thrust::sequence(thrust::device, deviceStructures.newVertexCommunity, deviceStructures.newVertexCommunity + V, 0);
	thrust::sequence(thrust::device, deviceStructures.originalToCommunity, deviceStructures.originalToCommunity + V, 0);

	HANDLE_ERROR(hipMemcpy(deviceStructures.communityWeight, hostStructures.communityWeight, V * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edges, hostStructures.edges, E * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.weights, hostStructures.weights, E * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edgesIndex, hostStructures.edgesIndex, (V + 1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.V, &hostStructures.V, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.E, &hostStructures.E, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.originalV, &hostStructures.originalV, sizeof(int), hipMemcpyHostToDevice));

	// preparing aggregationPhaseStructures
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.communityDegree, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newID, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgePos, V * sizeof(int)));;
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.vertexStart, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.orderedVertices, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgeIndexToCurPos, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newEdges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newWeights, E * sizeof(float)));
}

void copyStructuresWithInitPartition(host_structures& hostStructures, device_structures& deviceStructures,
					aggregation_phase_structures& aggregationPhaseStructures, int* comm_size) {
	// copying from deviceStructures to hostStructures
	int V = hostStructures.V, E = hostStructures.E;
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communityWeight, V * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.weights, E * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edgesIndex, (V + 1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalToCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexEdgesSum, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.newVertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.V, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.E, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalV, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communitySize, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.partition, V * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&deviceStructures.toOwnCommunity, V * sizeof(int)));


	// thrust::fill(thrust::device, deviceStructures.communitySize, deviceStructures.communitySize + V, 1);
	// thrust::sequence(thrust::device, deviceStructures.vertexCommunity, deviceStructures.vertexCommunity + V, 0);
	// thrust::sequence(thrust::device, deviceStructures.newVertexCommunity, deviceStructures.newVertexCommunity + V, 0);
	// thrust::sequence(thrust::device, deviceStructures.originalToCommunity, deviceStructures.originalToCommunity + V, 0);

	printf("Copy with init...\n");

	HANDLE_ERROR(hipMemcpy(deviceStructures.communitySize, comm_size, V * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.vertexCommunity, hostStructures.vertexCommunity, V * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.newVertexCommunity, hostStructures.vertexCommunity, V * sizeof(float), hipMemcpyHostToDevice));
	// HANDLE_ERROR(hipMemcpy(deviceStructures.originalToCommunity, hostStructures.vertexCommunity, V * sizeof(float), hipMemcpyHostToDevice));
	thrust::sequence(thrust::device, deviceStructures.originalToCommunity, deviceStructures.originalToCommunity + V, 0);

	HANDLE_ERROR(hipMemcpy(deviceStructures.communityWeight, hostStructures.communityWeight, V * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edges, hostStructures.edges, E * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.weights, hostStructures.weights, E * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edgesIndex, hostStructures.edgesIndex, (V + 1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.V, &hostStructures.V, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.E, &hostStructures.E, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.originalV, &hostStructures.originalV, sizeof(int), hipMemcpyHostToDevice));

	// preparing aggregationPhaseStructures
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.communityDegree, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newID, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgePos, V * sizeof(int)));;
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.vertexStart, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.orderedVertices, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgeIndexToCurPos, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newEdges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newWeights, E * sizeof(float)));
}

void deleteStructures(host_structures& hostStructures, device_structures& deviceStructures,
					  aggregation_phase_structures& aggregationPhaseStructures) {
    HANDLE_ERROR(hipHostFree(hostStructures.vertexCommunity));
    HANDLE_ERROR(hipHostFree(hostStructures.communityWeight));
    HANDLE_ERROR(hipHostFree(hostStructures.edges));
    HANDLE_ERROR(hipHostFree(hostStructures.weights));
    HANDLE_ERROR(hipHostFree(hostStructures.edgesIndex));
    HANDLE_ERROR(hipHostFree(hostStructures.originalToCommunity));


	HANDLE_ERROR(hipFree(deviceStructures.originalV));
    HANDLE_ERROR(hipFree(deviceStructures.vertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.communityWeight));
	HANDLE_ERROR(hipFree(deviceStructures.edges));
	HANDLE_ERROR(hipFree(deviceStructures.weights));
	HANDLE_ERROR(hipFree(deviceStructures.edgesIndex));
	HANDLE_ERROR(hipFree(deviceStructures.originalToCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.vertexEdgesSum));
	HANDLE_ERROR(hipFree(deviceStructures.newVertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.E));
	HANDLE_ERROR(hipFree(deviceStructures.V));
	HANDLE_ERROR(hipFree(deviceStructures.communitySize));
	HANDLE_ERROR(hipFree(deviceStructures.partition));
    HANDLE_ERROR(hipFree(deviceStructures.toOwnCommunity));

	HANDLE_ERROR(hipFree(aggregationPhaseStructures.communityDegree));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.newID));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.edgePos));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.vertexStart));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.orderedVertices));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.edgeIndexToCurPos));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.newEdges));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.newWeights));
}

int blocksNumber(int V, int threadsPerVertex) {
	return (V * threadsPerVertex + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}

bool isPrime(int n) {
	for (int i = 2; i < sqrt(n) + 1; i++)
		if (n % i == 0)
			return false;
	return true;
}

int getPrime(int n) {
	do {
		n++;
	} while(!isPrime(n));
	return n;
}

void parseCommandLineArgs(int argc, char *argv[], float *minGain, bool *isVerbose, char **fileName, char **initCommFileName, char **nodeEvalSetFileName) {
	bool isF, isG;
	char opt;
	while ((opt = getopt(argc, argv, "f:g:p:e:v")) != -1) {
		switch (opt) {
			case 'g':
				isG = true;
				*minGain = strtof(optarg, NULL);
				break;
			case 'v':
				*isVerbose = true;
				break;
			case 'f':
				isF = true;
				*fileName = optarg;
				break;
			case 'p':
				*initCommFileName = optarg;
				break;
			case 'e':
				*nodeEvalSetFileName = optarg;
				break;
			default:
				printf("Usage: ./gpulouvain -f mtx-matrix-file -g min-gain [-v]\n");
				exit(1);
		}
	}
	if (!isF || !isG) {
		printf("Usage: ./gpulouvain -f mtx-matrix-file -g min-gain [-v]\n");
		exit(1);
	}
}